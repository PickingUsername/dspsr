#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2016 by Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/TimeSeriesCUDA.h"
#include "dsp/MemoryCUDA.h"

#include "Error.h"

void check_error_stream (const char*, hipStream_t);

using namespace std;

void check_error (const char*);

__global__ void copy_data_fpt_kernel_ndim2(float2 * out, float2* in,
                                     uint64_t ichanpol_stride,
                                     uint64_t ochanpol_stride,
                                     uint64_t ndat)
{
  uint64_t dx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dx >= ndat)
    return;
  out[blockIdx.y * ochanpol_stride + dx] = in[blockIdx.y * ichanpol_stride + dx];
}

__global__ void copy_data_fpt_kernel_ndim1(float * out, float* in,
                                     uint64_t ichanpol_stride,
                                     uint64_t ochanpol_stride,
                                     uint64_t ndat)
{
  uint64_t dx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dx >= ndat)
    return;
  //if ((in[blockIdx.y * ichanpol_stride + dx] < -1) || (in[blockIdx.y * ichanpol_stride + dx] > 1))
  //  printf("[%d][%lu] %f\n", blockIdx.y, dx,  in[blockIdx.y * ichanpol_stride + dx]);
  out[blockIdx.y * ochanpol_stride + dx] = in[blockIdx.y * ichanpol_stride + dx];
}


template<typename T>
__global__ void copy_data_fpt_kernel(T * out, T* in,
                                     uint64_t ichanpol_stride,
                                     uint64_t ochanpol_stride,
                                     uint64_t ndat)
{
  uint64_t dx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dx >= ndat)
    return;
  out[blockIdx.y * ochanpol_stride + dx] = in[blockIdx.y * ichanpol_stride + dx];
}

CUDA::TimeSeriesEngine::TimeSeriesEngine (dsp::Memory * _memory)
{
  memory = dynamic_cast<CUDA::DeviceMemory*>(_memory);
  buffer = NULL;
  buffer_size = 0;
}

CUDA::TimeSeriesEngine::~TimeSeriesEngine ()
{
  if (buffer)
    memory->do_free (buffer);
  buffer = 0;
}

void CUDA::TimeSeriesEngine::prepare (dsp::TimeSeries * parent)
{
  to = parent;
}

void CUDA::TimeSeriesEngine::prepare_buffer (unsigned nbytes)
{
  if (nbytes > buffer_size)
  {
    if (buffer)
      memory->do_free (buffer);
    buffer_size = nbytes;
    buffer = memory->do_allocate (buffer_size);
    memory->do_zero(buffer, buffer_size);
  }
}

// copy data from another time series to this time series
void CUDA::TimeSeriesEngine::copy_data_fpt (const dsp::TimeSeries* from, 
    uint64_t idat_start, uint64_t ndat)
{
  // cuda device that is executing this function
  int device;
  hipGetDevice(&device);

#ifdef _DEBUG
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt from=" << (void *) from 
       << " idat_start=" << idat_start << " ndat=" << ndat << " device=" << device << endl;
#endif

  // stream and device upon which to TSE exists
  hipStream_t to_stream = memory->get_stream();
  int to_device          = memory->get_device();

  // stream and device upon which from TSE exists
  const CUDA::DeviceMemory * from_mem = dynamic_cast<const CUDA::DeviceMemory*>( from->get_memory());
  hipStream_t from_stream = from_mem->get_stream();
  const int from_device    = from_mem->get_device();

  if (!from_mem)
    throw Error (FailedSys, "CUDA::TimeSeriesEngine::copy_data_fpt", "From TimeSeries did not use DeviceMemory");

  unsigned nchan = from->get_nchan();
  unsigned npol  = from->get_npol();
  unsigned ndim  = from->get_ndim();
  
  uint64_t ichanpol_stride = 0;
  uint64_t ochanpol_stride = 0;
  uint64_t bchanpol_stride = ndat;

  if (npol > 1)
  {
    ochanpol_stride = to->get_datptr (0,1) - to->get_datptr (0,0);
    ichanpol_stride = from->get_datptr (0,1) - from->get_datptr (0,0);
  }
  else if (nchan > 1)
  {
    ochanpol_stride = to->get_datptr (1,0) - to->get_datptr (1,0);
    ichanpol_stride = from->get_datptr (1,0) - from->get_datptr (1,0);
  }
  else
  {
    ; 
  }

  ichanpol_stride /= ndim;
  ochanpol_stride /= ndim;

#ifdef _DEBUG
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt streams to="
       << (void*) to_stream << " from=" << (void*) from_stream << endl;
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt device to=" << device 
       << " from=" << from_device << endl;
  cerr  << "CUDA::TimeSeriesEngine::copy_data_fpt nchan=" << nchan << " ndim=" << ndim << " npol=" << npol << " ndat=" << ndat << endl;
  cerr  << "CUDA::TimeSeriesEngine::copy_data_fpt istride=" << ichanpol_stride << " ostride=" << ochanpol_stride << " bstride=" << bchanpol_stride << endl;
#endif

  unsigned nthread = 1024;
  if (nthread > ndat)
    nthread = ndat;
  dim3 blocks = dim3 (ndat / nthread, nchan*npol);
  if (ndat % nthread)
    blocks.x++;

#ifdef _DEBUG
  cerr << "blocks=(" << blocks.x << "," << blocks.y << ") threads=" << nthread << endl;
#endif

  size_t nbytes = nchan * ndim * npol * ndat * sizeof(float);

  // to un-stride from to the buffer
  if (to_stream != from_stream)
  {
    // now ensure the to TSE is of sufficient size
    if (device != to_device)
      hipSetDevice (to_device);
    prepare_buffer (nbytes);

    // switch to the from_device to ensure buffer is allocated
    if (to_device != from_device)
      hipSetDevice (from_device);

    CUDA::TimeSeriesEngine * from_engine = dynamic_cast<CUDA::TimeSeriesEngine*>(from->get_engine());
    from_engine->prepare_buffer (nbytes);

    hipStreamSynchronize (from_stream);

    // copy from -> buffer
    if (ndim == 2)
    {
      float2 * to_ptr   = (float2 *) from_engine->buffer;
      float2 * from_ptr = (float2 *) from->get_datptr (0,0);
      copy_data_fpt_kernel<float2><<<blocks,nthread,0,from_stream>>> (
        to_ptr, from_ptr + idat_start, ichanpol_stride, bchanpol_stride, ndat);
    }
    else
    {
      float * to_ptr   = (float *) from_engine->buffer;
      float * from_ptr = (float *) from->get_datptr (0,0);
      copy_data_fpt_kernel<float><<<blocks,nthread,0,from_stream>>> (
        to_ptr, from_ptr + idat_start, ichanpol_stride, bchanpol_stride, ndat);
      //hipDeviceSynchronize();
    }

    if (to_device != from_device)
    {
      hipMemcpyPeer (buffer, device, from_engine->buffer, to_device, nbytes);
    }
    else
    {
      // wait for the from stream to complete all pending work
      hipMemcpyAsync(buffer, from_engine->buffer, nbytes, hipMemcpyDeviceToDevice, from_stream);
      hipStreamSynchronize(from_stream);
    }

    // switch to the from_device to ensure buffer is allocated
    if (to_device != from_device)
      hipSetDevice (to_device);

    hipStreamSynchronize(to_stream);

    // copy buffer -> to
    if (ndim == 2)
    {
      float2 * to_ptr   = (float2 *) to->get_datptr (0,0);
      float2 * from_ptr = (float2 *) buffer;
      copy_data_fpt_kernel_ndim2<<<blocks,nthread,0,to_stream>>> (
          to_ptr, from_ptr, bchanpol_stride, ochanpol_stride, ndat);
    }
    else
    {
      float * to_ptr   = (float *) to->get_datptr (0,0);
      float * from_ptr = (float *) buffer;
      copy_data_fpt_kernel_ndim1<<<blocks,nthread,0,to_stream>>> (
        to_ptr, from_ptr, bchanpol_stride, ochanpol_stride, ndat);
      //hipDeviceSynchronize();
    }
    if (device != to_device || device != from_device)
      hipSetDevice(device);
    hipStreamSynchronize(to_stream);
  }
  // in the same stream & device
  else
  {
    if (ndim == 2)
    {
      float2 * to_ptr   = (float2 *) to->get_datptr (0,0);
      float2 * from_ptr = (float2 *) from->get_datptr (0,0);
      copy_data_fpt_kernel_ndim2<<<blocks,nthread,0,to_stream>>> (
        to_ptr, from_ptr + idat_start, ichanpol_stride, ochanpol_stride, ndat);
    }
    else
    {
      float * to_ptr   = (float *) to->get_datptr (0,0);
      float * from_ptr = (float *) from->get_datptr (0,0);
      copy_data_fpt_kernel_ndim1<<<blocks,nthread,0,to_stream>>> (
        to_ptr, from_ptr + idat_start, ichanpol_stride, ochanpol_stride, ndat);
      //hipDeviceSynchronize();
    }
  }
}

