#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2016 by Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/TimeSeriesCUDA.h"
#include "dsp/MemoryCUDA.h"

#include "Error.h"

void check_error_stream (const char*, hipStream_t);

using namespace std;

void check_error (const char*);

template<typename T>
__global__ void copy_data_fpt_kernel(T * out, T* in,
                                     uint64_t ichanpol_stride,
                                     uint64_t ochanpol_stride,
                                     uint64_t ndat)
{
  uint64_t dx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dx >= ndat)
    return;
  out[blockIdx.y * ochanpol_stride + dx] = in[blockIdx.y * ichanpol_stride + dx];
}

CUDA::TimeSeriesEngine::TimeSeriesEngine (dsp::Memory * _memory)
{
  memory = dynamic_cast<CUDA::DeviceMemory*>(_memory);
  buffer = NULL;
  buffer_size = 0;
}

CUDA::TimeSeriesEngine::~TimeSeriesEngine ()
{
  if (buffer)
    memory->do_free (buffer);
  buffer = 0;
}

void CUDA::TimeSeriesEngine::prepare (dsp::TimeSeries * parent)
{
  to = parent;
}

void CUDA::TimeSeriesEngine::prepare_buffer (unsigned nbytes)
{
  if (nbytes > buffer_size)
  {
    if (buffer)
      memory->do_free (buffer);
    buffer_size = nbytes;
    buffer = memory->do_allocate (buffer_size);
  }
}

// copy data from another time series to this time series
void CUDA::TimeSeriesEngine::copy_data_fpt (const dsp::TimeSeries* from, 
    uint64_t idat_start, uint64_t ndat)
{
  // cuda device that is executing this function
  int device;
  hipGetDevice(&device);
  
#ifdef _DEBUG
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt from=" << (void *) from 
       << " idat_start=" << idat_start << " ndat=" << ndat << " device=" << device << endl;
#endif

  // stream and device upon which to TSE exists
  hipStream_t to_stream = memory->get_stream();
  int to_device          = memory->get_device();

  // stream and device upon which from TSE exists
  const CUDA::DeviceMemory * from_mem = dynamic_cast<const CUDA::DeviceMemory*>( from->get_memory());
  hipStream_t from_stream = from_mem->get_stream();
  const int from_device    = from_mem->get_device();

  if (!from_mem)
    throw Error (FailedSys, "CUDA::TimeSeriesEngine::copy_data_fpt", "From TimeSeries did not use DeviceMemory");

  unsigned nchan = to->get_nchan();
  unsigned npol  = to->get_npol();
  unsigned ndim  = to->get_ndim();
  
  uint64_t ichanpol_stride = 0;
  uint64_t ochanpol_stride = 0;
  uint64_t bchanpol_stride = ndat;

  if (npol > 1)
  {
    ochanpol_stride = to->get_datptr (0,1) - to->get_datptr (0,0);
    ichanpol_stride = from->get_datptr (0,1) - from->get_datptr (0,0);
  }
  else if (nchan > 1)
  {
    ochanpol_stride = to->get_datptr (1,0) - to->get_datptr (1,0);
    ichanpol_stride = from->get_datptr (1,0) - from->get_datptr (1,0);
  }
  else
  {
    ; 
  }

  ichanpol_stride /= ndim;
  ochanpol_stride /= ndim;

#ifdef _DEBUG
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt streams to="
       << (void*) to_stream << " from=" << (void*) from_stream << endl;
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt device to=" << device 
       << " from=" << from_device << endl;
#endif

  unsigned nthread = 1024;
  if (nthread > ndat)
    nthread = ndat;
  dim3 blocks = dim3 (ndat / nthread, nchan*npol);
  if (ndat % nthread)
    blocks.x++;

  size_t nbytes = nchan * ndim * npol * ndat * sizeof(float);

  // to un-stride from to the buffer
  if (to_stream != from_stream)
  {
    // now ensure the to TSE is of sufficient size
    if (device != to_device)
      hipSetDevice (to_device);
    prepare_buffer (nbytes);

    // switch to the from_device to ensure buffer is allocated
    if (to_device != from_device)
      hipSetDevice (from_device);

    CUDA::TimeSeriesEngine * from_engine = dynamic_cast<CUDA::TimeSeriesEngine*>(from->get_engine());

    from_engine->prepare_buffer (nbytes);

    // copy from -> buffer
    if (ndim == 2)
    {
      float2 * to_ptr   = (float2 *) from_engine->buffer;
      float2 * from_ptr = (float2 *) from->get_datptr (0,0);
      copy_data_fpt_kernel<float2><<<blocks,nthread,0,from_stream>>> (
        to_ptr, from_ptr + idat_start, ichanpol_stride, bchanpol_stride, ndat);
    }
    else
    {
      float * to_ptr   = (float *) from_engine->buffer;
      float * from_ptr = (float *) from->get_datptr (0,0);
      copy_data_fpt_kernel<float><<<blocks,nthread,0,from_stream>>> (
        to_ptr, from_ptr + idat_start, ichanpol_stride, bchanpol_stride, ndat);
    }

    if (to_device != from_device)
    {
      hipMemcpyPeer (buffer, device, from_engine->buffer, to_device, nbytes);
    }
    else
    {
      hipMemcpy(buffer, from_engine->buffer, nbytes, hipMemcpyDeviceToDevice);
    }

    // switch to the from_device to ensure buffer is allocated
    if (to_device != from_device)
      hipSetDevice (to_device);

    // copy buffer -> to
    if (ndim == 2)
    {
      float2 * to_ptr   = (float2 *) to->get_datptr (0,0);
      float2 * from_ptr = (float2 *) buffer;
      copy_data_fpt_kernel<float2><<<blocks,nthread,0,to_stream>>> (
          to_ptr, from_ptr, bchanpol_stride, ochanpol_stride, ndat);
    }
    else
    {
      float * to_ptr   = (float *) to->get_datptr (0,0);
      float * from_ptr = (float *) buffer;
      copy_data_fpt_kernel<float><<<blocks,nthread,0,to_stream>>> (
        to_ptr, from_ptr, bchanpol_stride, ochanpol_stride, ndat);
    }
  }
  // in the same stream & device
  else
  {
    if (ndim == 2)
    {
      float2 * to_ptr   = (float2 *) to->get_datptr (0,0);
      float2 * from_ptr = (float2 *) from->get_datptr (0,0);
      copy_data_fpt_kernel<float2><<<blocks,nthread,0,to_stream>>> (
        to_ptr, from_ptr + idat_start, ichanpol_stride, ochanpol_stride, ndat);
    }
    else
    {
      float * to_ptr   = (float *) to->get_datptr (0,0);
      float * from_ptr = (float *) from->get_datptr (0,0);
      copy_data_fpt_kernel<float><<<blocks,nthread,0,to_stream>>> (
        to_ptr, from_ptr + idat_start, ichanpol_stride, ochanpol_stride, ndat);
    }
  }

  hipSetDevice(device);
}

