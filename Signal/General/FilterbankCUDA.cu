#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2010 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

// #define _DEBUG 1

#include "dsp/FilterbankCUDA.h"
#include "debug.h"

#include <hip/hip_runtime.h>

#include <iostream>
#include <assert.h>

void check_error (const char*);

#ifdef _DEBUG
#define CHECK_ERROR(x) check_error(x)
#else
#define CHECK_ERROR(x)
#endif


// compute 2X(w) = Z(w) + Z^*(-w) 
#define sep_X(X,z,zh) X.x = 0.5*(z.x + zh.x); X.y = 0.5*(z.y - zh.y);

// compute 2Y(w) = iZ^*(-w) - iZ(w)
#define sep_Y(Y,z,zh) Y.x = 0.5*(zh.y + z.y); Y.y = 0.5*(zh.x - z.x);

__global__ void separate (float2* d_fft, int nfft)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
  int k = nfft - i;

  float2* p0 = d_fft;
  float2* p1 = d_fft + nfft;

  float2 p0i = p0[i];
  float2 p0k = p0[k];

  float2 p1i = p1[i];
  float2 p1k = p1[k];

  sep_X( p0[i], p0i, p1k );
  sep_X( p0[k], p0k, p1i );

  sep_Y( p1[i], p0i, p1k );
  sep_Y( p1[k], p0k, p1i );
}

__global__ void multiply (float2* d_fft, float2* kernel)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  float x = d_fft[i].x * kernel[i].x - d_fft[i].y * kernel[i].y;
  d_fft[i].y = d_fft[i].x * kernel[i].y + d_fft[i].y * kernel[i].x;
  d_fft[i].x = x;
}

__global__ void ncopy (float2* output_data, unsigned output_stride,
           const float2* input_data, unsigned input_stride,
           unsigned to_copy)
{
  output_data += blockIdx.y * output_stride;
  input_data += blockIdx.y * input_stride;

  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < to_copy)
    output_data[index] = input_data[index];
}


using namespace std;

CUDA::FilterbankEngine::FilterbankEngine (hipStream_t _stream)
{
  real_to_complex = false;

  d_fft = d_kernel = 0;

  stream = _stream;

  plan_fwd = 0;
  plan_bwd = 0;
  verbose = false;
}

CUDA::FilterbankEngine::~FilterbankEngine ()
{
}

void CUDA::FilterbankEngine::setup (dsp::Filterbank* filterbank)
{
  freq_res = filterbank->get_freq_res ();
  nchan_subband = filterbank->get_nchan_subband();

  real_to_complex = (filterbank->get_input()->get_state() == Signal::Nyquist);

  DEBUG("CUDA::FilterbankEngine::setup nchan_subband=" << nchan_subband \
        << " freq_res=" << freq_res);

  DEBUG("CUDA::FilterbankEngine::setup scratch=" << scratch);

  // determine GPU capabilities 
  int device = 0;
  hipGetDevice(&device);
  struct hipDeviceProp_t device_properties;
  hipGetDeviceProperties (&device_properties, device);
  max_threads_per_block = device_properties.maxThreadsPerBlock;

  if (real_to_complex)
  {
    DEBUG("CUDA::FilterbankEngine::setup plan size=" << freq_res*nchan_subband*2);
    hipfftPlan1d (&plan_fwd, freq_res*nchan_subband*2, HIPFFT_R2C, 1);
  }
  else
  {
    DEBUG("CUDA::FilterbankEngine::setup plan size=" << freq_res*nchan_subband);
    hipfftPlan1d (&plan_fwd, freq_res*nchan_subband, HIPFFT_C2C, 1);
  }

  DEBUG("CUDA::FilterbankEngine::setup setting stream " << stream);
  hipfftSetStream (plan_fwd, stream);

  // optimal performance for CUFFT regarding data layout
  cufftSetCompatibilityMode(plan_fwd, CUFFT_COMPATIBILITY_NATIVE);

  DEBUG("CUDA::FilterbankEngine::setup fwd FFT plan set");
  if (freq_res > 1)
  {
    hipfftPlan1d (&plan_bwd, freq_res, HIPFFT_C2C, nchan_subband);
    hipfftSetStream (plan_bwd, stream);

    // optimal performance for CUFFT regarding data layout
    cufftSetCompatibilityMode(plan_bwd, CUFFT_COMPATIBILITY_NATIVE);
    DEBUG("CUDA::FilterbankEngine::setup bwd FFT plan set");
  }

  if (filterbank->has_response())
  {
    const dsp::Response* response = filterbank->get_response();

    unsigned nchan = response->get_nchan();
    unsigned ndat = response->get_ndat();
    unsigned ndim = response->get_ndim();

    assert( nchan == filterbank->get_nchan() );
    assert( ndat == freq_res );
    assert( ndim == 2 ); // complex

    unsigned mem_size = nchan * ndat * ndim * sizeof(hipfftReal);

    // allocate space for the convolution kernel
    hipMalloc ((void**)&d_kernel, mem_size);

    nfilt_pos = response->get_impulse_pos();
    unsigned nfilt_tot = nfilt_pos + response->get_impulse_neg();

    // points kept from each small fft
    nkeep = freq_res - nfilt_tot;
 
    // copy the kernel accross
    const float* kernel = filterbank->get_response()->get_datptr(0,0);

    hipMemcpy (d_kernel, kernel, mem_size, hipMemcpyHostToDevice);
  }

  if (!real_to_complex)
    return;
}

void CUDA::FilterbankEngine::set_scratch (float * _scratch)
{
  scratch = _scratch;
}

extern void check_error (const char*);

void CUDA::FilterbankEngine::finish ()
{
  check_error ("CUDA::FilterbankEngine::finish");
}


void CUDA::FilterbankEngine::perform (const dsp::TimeSeries * in, dsp::TimeSeries * out, 
            uint64_t npart, const uint64_t in_step, const uint64_t out_step)
{
  verbose = dsp::Operation::record_time || dsp::Operation::verbose;

  const unsigned npol = in->get_npol();
  const unsigned input_nchan = in->get_nchan();
  const unsigned output_nchan = out->get_nchan();
 
  // counters
  unsigned ipol, ichan;
  uint64_t ipart;
 
  // offsets into input and output
  uint64_t in_offset, out_offset;

  // GPU scratch space
  DEBUG("CUDA::FilterbankEngine::perform scratch=" << scratch);
  float2* cscratch = (float2*) scratch;

  unsigned data_size = nchan_subband * freq_res;
  int threads_per_block = max_threads_per_block / 2;

  // note that each thread will set two complex numbers in each poln
  int blocks = data_size / (threads_per_block * 2);

  float * output_ptr;
  float * input_ptr;
  uint64_t output_span;

  DEBUG("CUDA::FilterbankEngine::perform input_nchan=" << input_nchan);
  DEBUG("CUDA::FilterbankEngine::perform npol=" << npol);
  DEBUG("CUDA::FilterbankEngine::perform npart=" << npart);
  DEBUG("CUDA::FilterbankEngine::perform nkeep=" << nkeep);
  DEBUG("CUDA::FilterbankEngine::perform in_step=" << in_step);
  DEBUG("CUDA::FilterbankEngine::perform out_step=" << out_step);

  for (ichan=0; ichan<input_nchan; ichan++)
  {
    for (ipol=0; ipol < npol; ipol++)
    {
      for (ipart=0; ipart < npart; ipart++)
      {
        DEBUG("CUDA::FilterbankEngine::perform ipart " << ipart << " of " << npart);

        in_offset = ipart * in_step;
        out_offset = ipart * out_step;

        //DEBUG("CUDA::FilterbankEngine::perform offsets in=" << in_offset << " out=" << out_offset);

        input_ptr = const_cast<float*>(in->get_datptr (ichan, ipol)) + in_offset;

        //DEBUG("CUDA::FilterbankEngine::perform FORWARD FFT");
        if (real_to_complex)
        {
          hipfftExecR2C(plan_fwd, input_ptr, cscratch);
          check_error ("CUDA::FilterbankEngine::perform hipfftExecR2C FORWARD");
        }
        else
        {
          float2* cin = (float2*) input_ptr;
          hipfftExecC2C(plan_fwd, cin, cscratch, HIPFFT_FORWARD);
          check_error ("CUDA::FilterbankEngine::perform hipfftExecC2C FORWARD");
        }

        blocks = data_size / threads_per_block;

        if (d_kernel)
        {
          DEBUG("CUDA::FilterbankEngine::perform multiply dedipersion kernel");
          multiply<<<blocks,threads_per_block,0,stream>>> (cscratch, d_kernel);
          check_error ("CUDA::FilterbankEngine::perform multiply");
        }

        //DEBUG("CUDA::FilterbankEngine::perform BACKWARD FFT");
        hipfftExecC2C (plan_bwd, cscratch, cscratch, HIPFFT_BACKWARD);

        check_error ("CUDA::FilterbankEngine::perform hipfftExecC2C BACKWARD");

        if (out)
        {
          output_ptr = out->get_datptr (0, ipol) + out_offset;
          output_span = out->get_datptr (1, ipol) - out->get_datptr (0, ipol);

          const float2* input = cscratch + nfilt_pos;
          unsigned input_stride = freq_res;
          unsigned to_copy = nkeep;

          {
            dim3 threads;
            threads.x = threads_per_block;

            dim3 blocks;
            blocks.x = nkeep / threads.x;
            if (nkeep % threads.x)
              blocks.x ++;

            blocks.y = nchan_subband;

            // divide by two for complex data
            float2* output_base = (float2*) output_ptr;
            unsigned output_stride = output_span / 2;

            DEBUG("CUDA::FilterbankEngine::perform output base=" << output_base << " stride=" << output_stride);
            DEBUG("CUDA::FilterbankEngine::perform input base=" << input << " stride=" << input_stride);
            DEBUG("CUDA::FilterbankEngine::perform to_copy=" << to_copy);

            ncopy<<<blocks,threads,0,stream>>> (output_base, output_stride,
                        input, input_stride, to_copy);
            check_error ("CUDA::FilterbankEngine::perform ncopy");
          }
        } // if not benchmarking
      } // for each part
    } // for each polarization
  } // for each channel

  if (verbose)
    check_error ("CUDA::FilterbankEngine::perform");
}
