#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2011 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

// #define _DEBUG 1

#include "dsp/filterbank_engine.h"
#include "dsp/filterbank_cuda.h"
#include "debug.h"

void check_error (const char*);

#ifdef _DEBUG
#define CHECK_ERROR(x) check_error(x)
#else
#define CHECK_ERROR(x)
#endif
/* *************************************************************************
 *
 *
 * The twofft trick
 *
 * Where:
 *   Z = X + i Y
 *   X, Y, and Z are complex
 *   X(-w) = X*(w)
 *   Y(-w) = X*(w)
 *   Z^*(-w) = X(w) - i Y(w)
 *
 *
 ************************************************************************* */

// compute 2X(w) = Z(w) + Z^*(-w) 
#define sep_X(X,z,zh) X.x = 0.5*(z.x + zh.x); X.y = 0.5*(z.y - zh.y);

// compute 2Y(w) = iZ^*(-w) - iZ(w)
#define sep_Y(Y,z,zh) Y.x = 0.5*(zh.y + z.y); Y.y = 0.5*(zh.x - z.x);

__global__ void separate (float2* d_fft, int nfft)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
  int k = nfft - i;

  float2* p0 = d_fft;
  float2* p1 = d_fft + nfft;

  float2 p0i = p0[i];
  float2 p0k = p0[k];

  float2 p1i = p1[i];
  float2 p1k = p1[k];

  sep_X( p0[i], p0i, p1k );
  sep_X( p0[k], p0k, p1i );

  sep_Y( p1[i], p0i, p1k );
  sep_Y( p1[k], p0k, p1i );
}

__global__ void multiply (float2* d_fft, float2* kernel)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  float x = d_fft[i].x * kernel[i].x - d_fft[i].y * kernel[i].y;
  d_fft[i].y = d_fft[i].x * kernel[i].y + d_fft[i].y * kernel[i].x;
  d_fft[i].x = x;
}

__global__ void ncopy (float2* output_data, unsigned output_stride, 
		       const float2* input_data, unsigned input_stride,
		       unsigned to_copy)
{
  output_data += blockIdx.y * output_stride;
  input_data += blockIdx.y * input_stride;

  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < to_copy)
    output_data[index] = input_data[index];
}

void filterbank_cuda_perform (filterbank_engine* engine, 
			      filterbank_cuda* cuda,
			      const float* in, 
            const int max_threads_per_block)
{
  float2* cscratch = (float2*) engine->scratch;

  unsigned data_size = engine->nchan * cuda->bwd_nfft;
  int threads_per_block = max_threads_per_block / 2;

  // note that each thread will set two complex numbers in each poln
  int blocks = data_size / (threads_per_block * 2);

  if (in)
  {
    if (cuda->real_to_complex)
    {
      float * cin = (float *) in;
      hipfftExecR2C(cuda->plan_fwd, cin, cscratch);
      CHECK_ERROR ("CUDA::FilterbankEngine::perform hipfftExecR2C FORWARD");
    }
    else
    {
      float2* cin = (float2*) in;
      hipfftExecC2C(cuda->plan_fwd, cin, cscratch, HIPFFT_FORWARD);
      CHECK_ERROR ("CUDA::FilterbankEngine::perform hipfftExecR2C FORWARD");
    }
  }

  blocks = data_size / threads_per_block;

  if (cuda->d_kernel)
  {
    multiply<<<blocks,threads_per_block,0,cuda->stream>>> (cscratch, cuda->d_kernel);
    CHECK_ERROR ("CUDA::FilterbankEngine::perform multiply");
  }

  hipfftExecC2C (cuda->plan_bwd, cscratch, cscratch, HIPFFT_BACKWARD);

  CHECK_ERROR ("CUDA::FilterbankEngine::perform hipfftExecC2C BACKWARD");

  if (!engine->output)
    return;

  const float2* input = cscratch + engine->nfilt_pos;
  unsigned input_stride = cuda->bwd_nfft;
  unsigned to_copy = engine->nkeep;

  {
    dim3 threads;
    threads.x = threads_per_block;

    dim3 blocks;
    blocks.x = engine->nkeep / threads.x;
    if (engine->nkeep % threads.x)
      blocks.x ++;

    blocks.y = engine->nchan;
    
    // divide by two for complex data
    float2* output_base = (float2*) engine->output;
    unsigned output_stride = engine->output_span / 2;
    
    ncopy<<<blocks,threads,0,cuda->stream>>> (output_base, output_stride,
					      input, input_stride, to_copy);
  }
  
  if (cuda->verbose)
    check_error ("CUDA::FilterbankEngine::perform");
}
