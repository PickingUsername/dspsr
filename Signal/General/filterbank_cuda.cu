#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2011 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

// #define _DEBUG 1

#include "dsp/filterbank_engine.h"
#include "dsp/filterbank_cuda.h"
#include "debug.h"

void check_error (const char*);

#ifdef _DEBUG
#define CHECK_ERROR(x) check_error(x)
#else
#define CHECK_ERROR(x)
#endif
/* *************************************************************************
 *
 *
 * The twofft trick
 *
 * Where:
 *   Z = X + i Y
 *   X, Y, and Z are complex
 *   X(-w) = X*(w)
 *   Y(-w) = X*(w)
 *   Z^*(-w) = X(w) - i Y(w)
 *
 *
 ************************************************************************* */

// compute 2X(w) = Z(w) + Z^*(-w) 
#define sep_X(X,z,zh) X.x = 0.5*(z.x + zh.x); X.y = 0.5*(z.y - zh.y);

// compute 2Y(w) = iZ^*(-w) - iZ(w)
#define sep_Y(Y,z,zh) Y.x = 0.5*(zh.y + z.y); Y.y = 0.5*(zh.x - z.x);

__global__ void separate (float2* d_fft, int nfft)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
  int k = nfft - i;

  float2* p0 = d_fft;
  float2* p1 = d_fft + nfft;

  float2 p0i = p0[i];
  float2 p0k = p0[k];

  float2 p1i = p1[i];
  float2 p1k = p1[k];

  sep_X( p0[i], p0i, p1k );
  sep_X( p0[k], p0k, p1i );

  sep_Y( p1[i], p0i, p1k );
  sep_Y( p1[k], p0k, p1i );
}

/* *************************************************************************
 *
 *
 * The realtr trick
 *
 *
 ************************************************************************* */

__global__ void realtr (float2* d_fft, unsigned bwd_nfft,
			float* k_SN, float* k_CN)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int k = bwd_nfft - i;
 
  float real_aa=d_fft[i].x+d_fft[k].x;
  float real_ab=d_fft[k].x-d_fft[i].x;
  
  float imag_ba=d_fft[i].y+d_fft[k].y;
  float imag_bb=d_fft[k].y-d_fft[i].y;

  float temp_real=k_CN[i]*imag_ba+k_SN[i]*real_ab;
  float temp_imag=k_SN[i]*imag_ba-k_CN[i]*real_ab;

  d_fft[k].y = -0.5*(temp_imag-imag_bb);
  d_fft[i].y = -0.5*(temp_imag+imag_bb);

  d_fft[k].x = 0.5*(real_aa-temp_real);
  d_fft[i].x = 0.5*(real_aa+temp_real);
}

/* *************************************************************************
 *
 *
 * end of tricks
 *
 *
 ************************************************************************* */

__global__ void multiply (float2* d_fft, float2* kernel)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  float x = d_fft[i].x * kernel[i].x - d_fft[i].y * kernel[i].y;
  d_fft[i].y = d_fft[i].x * kernel[i].y + d_fft[i].y * kernel[i].x;
  d_fft[i].x = x;
}

__global__ void ncopy (float2* output_data, unsigned output_stride, 
		       const float2* input_data, unsigned input_stride,
		       unsigned to_copy)
{
  output_data += blockIdx.y * output_stride;
  input_data += blockIdx.y * input_stride;

  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < to_copy)
    output_data[index] = input_data[index];
}

void filterbank_cuda_perform (filterbank_engine* engine, 
			      filterbank_cuda* cuda,
			      const float* in)
{
  float2* cscratch = (float2*) engine->scratch;
  float2* cin = (float2*) in;

  unsigned data_size = engine->nchan * cuda->bwd_nfft;
  int threads = 256;

  // note that each thread will set two complex numbers in each poln
  int blocks = data_size / (threads*2);

  if (in)
  {
    hipfftExecC2C (cuda->plan_fwd, cin, cscratch, HIPFFT_FORWARD);

    CHECK_ERROR ("CUDA::FilterbankEngine::perform hipfftExecC2C FORWARD");

    if (engine->nchan == 1)
      return;

    if (cuda->real_to_complex)
    {
      DEBUG("CUDA::FilterbankEngine::perform real-to-complex");

      realtr<<<blocks,threads,0,cuda->stream>>> (cscratch,data_size,
					         cuda->d_SN,
					         cuda->d_CN);

      CHECK_ERROR ("CUDA::FilterbankEngine::perform realtr");
    }
  }

  blocks = data_size / threads;

  if (cuda->d_kernel)
  {
    multiply<<<blocks,threads,0,cuda->stream>>> (cscratch, cuda->d_kernel);
    CHECK_ERROR ("CUDA::FilterbankEngine::perform multiply");
  }

  hipfftExecC2C (cuda->plan_bwd, cscratch, cscratch, HIPFFT_BACKWARD);

  CHECK_ERROR ("CUDA::FilterbankEngine::perform hipfftExecC2C BACKWARD");

  if (!engine->output)
    return;

  const float2* input = cscratch + engine->nfilt_pos;
  unsigned input_stride = cuda->bwd_nfft;
  unsigned to_copy = engine->nkeep;

  {
    dim3 threads;
    threads.x = 128;

    dim3 blocks;
    blocks.x = engine->nkeep / threads.x;
    if (engine->nkeep % threads.x)
      blocks.x ++;

    blocks.y = engine->nchan;
    
    // divide by two for complex data
    float2* output_base = (float2*) engine->output;
    unsigned output_stride = engine->output_span / 2;
    
    ncopy<<<blocks,threads,0,cuda->stream>>> (output_base, output_stride,
					      input, input_stride, to_copy);
  }
  
  if (cuda->verbose)
    check_error ("CUDA::FilterbankEngine::perform");
}
