#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2010
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/FoldCUDA.h"

#include "Error.h"

#include <memory>

using namespace std;

CUDA::FoldEngine::FoldEngine ()
{
  d_bin = 0;
  d_bin_size = 0;
}

CUDA::FoldEngine::~FoldEngine ()
{
  if (d_bin)
    hipFree (d_bin);
}

void CUDA::FoldEngine::set_nbin (unsigned nbin)
{
  current_bin = folding_nbin = nbin;
  current_hits = 0;
  ndat_fold = 0;
  binplan.resize (0);
}

void CUDA::FoldEngine::set_bin (uint64_t idat, unsigned ibin)
{
  if (ibin != current_bin)
  {
    if (current_bin != folding_nbin)
      binplan.back().hits = current_hits;

    bin start;
    start.offset = idat;
    start.ibin = ibin;

    binplan.push_back ( start );

    current_bin = ibin;
    current_hits = 0;
  }
  ndat_fold ++;
  current_hits ++;
}

void CUDA::FoldEngine::send_binplan ()
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::send_binplan ndat=" << ndat_fold 
         << " intervals=" << binplan.size() << endl;

  uint64_t mem_size = binplan.size() * sizeof(bin);

  if (binplan.size() > d_bin_size)
  {
    if (d_bin)
      hipFree (d_bin);

    hipMalloc ((void**)&d_bin, mem_size);
    d_bin_size = binplan.size();
  }
 
  // copy the kernel accross
  hipError_t error;
  error = hipMemcpy (d_bin, &(binplan[0]), mem_size, hipMemcpyHostToDevice);
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::FoldEngine::set_binplan",
                 "this=%x %s", this, hipGetErrorString (error));
}


// header for if we decide to calculate weights on gpu too...
//__global__ void calculateWeight ()
//{
//  unsigned threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
//}


__global__ void fold1bin (const float* in_base,
			     unsigned in_span,
			     float* out_base,
			     unsigned out_span,
			     unsigned ndim,
			     unsigned nbin,
			     unsigned binplan_size,
			     CUDA::bin* binplan)
{
  unsigned ibin = blockIdx.x * blockDim.x + threadIdx.x;

  in_base += in_span * blockIdx.y + threadIdx.z;
  out_base += out_span * blockIdx.y + threadIdx.z;

  float total = 0;

  for (unsigned jbin=ibin; jbin < binplan_size; jbin += nbin)
  {
    const float* input = in_base + binplan[jbin].offset * ndim;

    for (unsigned i=0; i < binplan[jbin].hits; i++)
      total += input[i*ndim];
  }

  out_base[ binplan[ibin].ibin * ndim ] += total;
}

std::ostream& operator<< (std::ostream& ostr, const dim3& v)
{
  return ostr << "(" << v.x << "," << v.y << "," << v.z << ")";
}

void CUDA::FoldEngine::fold ()
{
  send_binplan ();

  dim3 blockDim (128, 1, ndim);
  dim3 gridDim (folding_nbin/128, npol*nchan, 1);

#if 0
  cerr << "blockDim=" << blockDim << endl;
  cerr << "gridDim=" << gridDim << endl;
#endif

  fold1bin<<<gridDim,blockDim>>> (input, input_span, output, output_span,
                                  ndim, folding_nbin, binplan.size(), d_bin);

  hipDeviceSynchronize ();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::FoldEngine::fold", 
                 hipGetErrorString (error));

}

