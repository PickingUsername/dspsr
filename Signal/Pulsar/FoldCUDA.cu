#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2010
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/FoldCUDA.h"

#include "Error.h"

#include <memory>

using namespace std;

CUDA::FoldEngine::FoldEngine ()
{
  d_bin = 0;
  d_bin_size = 0;
}

CUDA::FoldEngine::~FoldEngine ()
{
  if (d_bin)
    hipFree (d_bin);
}

void CUDA::FoldEngine::set_nbin (unsigned nbin)
{
  current_bin = folding_nbin = nbin;
  current_hits = 0;
  binplan.resize (0);
}

void CUDA::FoldEngine::set_bin (uint64_t idat, unsigned ibin)
{
  if (ibin != current_bin)
  {
    if (current_bin != folding_nbin)
      binplan.last().hits = current_hits;

    bin start;
    start.offset = idat;
    start.ibin = ibin;

    binplan.push_back ( start );

    current_bin = ibin;
    current_hits = 0;
  }

  current_hits ++;
}

void CUDA::FoldEngine::send_binplan ()
{
  uint64_t mem_size = binplan.size() * sizeof(bin);

  if (binplan.size() > d_bin_size)
  {
    if (d_bin)
      hipFree (d_bin);

    hipMalloc ((void**)&d_bin, mem_size);
    d_bin_size = binplan.size();
  }
 
  // copy the kernel accross
  hipError_t error;
  error = hipMemcpy (d_bin, &(binplan[0]), mem_size, hipMemcpyHostToDevice);
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::FoldEngine::set_binplan",
                 "this=%x %s", this, hipGetErrorString (error));
}


// header for if we decide to calculate weights on gpu too...
//__global__ void calculateWeight ()
//{
//  unsigned threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
//}


__global__ void performFold (const float* in_base,
			     unsigned in_span,
			     float* out_base,
			     unsigned out_span,
			     unsigned ndim,
			     unsigned nbin,
			     unsigned binplan_size,
			     bin* binplan)
{
  unsigned ibin = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned iblock = blockIdx.y;
  unsigned idim = threadIdx.z;

  in_base += in_span * iblock + idim;
  out_base += out_span * iblock + idim;

  float total = 0;

  for (unsigned jbin=ibin; jbin < binplan_size; jbin += nbin)
  {
    float* input = in_base + binplan[jbin].offset * ndim;

    for (unsigned i=0; i < binplan[jbin].hits; i++)
      total += input[i*ndim];
  }

  out_base[ibin*ndim] += total;
}

std::ostream& operator<< (std::ostream& ostr, const dim3& v)
{
  return ostr << "(" << v.x << "," << v.y << "," << v.z << ")";
}

void CUDA::FoldEngine::fold ()
{
  send_binplan ();

  dim3 blockDim (128, 1, ndim);
  dim3 gridDim (folding_nbin/128, npol*nchan, 1);

#if 0
  cerr << "blockDim=" << blockDim << endl;
  cerr << "gridDim=" << gridDim << endl;
#endif

  add_bin<<<gridDim,blockDim>>> (input, input_span,
				 output, output_span,
				 ndim,
				 nbin, binplan.size(), d_bin);

  hipDeviceSynchronize ();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::FoldEngine::fold", 
                 hipGetErrorString (error));

}

