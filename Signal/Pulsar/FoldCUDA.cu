#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2010 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/FoldCUDA.h"
#include "dsp/MemoryCUDA.h"

#include "Error.h"

#include <memory>

using namespace std;

CUDA::FoldEngine::FoldEngine ()
{
  d_bin = 0;
  d_bin_size = 0;

  d_profiles = new dsp::PhaseSeries;
  d_profiles->set_memory( new CUDA::DeviceMemory );

  // no data on either the host or device
  synchronized = true;
}

CUDA::FoldEngine::~FoldEngine ()
{
  if (d_bin)
    hipFree (d_bin);
}

void CUDA::FoldEngine::set_nbin (unsigned nbin)
{
  current_bin = folding_nbin = nbin;
  current_hits = 0;
  ndat_fold = 0;
  binplan.resize (0);
}

void CUDA::FoldEngine::set_bin (uint64_t idat, unsigned ibin)
{
  if (ibin != current_bin)
  {
    /* store the number of time samples to integrate
       in the interval that just ended */
    if (binplan.size())
      binplan.back().hits = current_hits;

    bin start;
    start.offset = idat;
    start.ibin = ibin;

    /* start a new interval */
    binplan.push_back ( start );

    current_bin = ibin;
    current_hits = 0;
  }

  ndat_fold ++;
  current_hits ++;
}

dsp::PhaseSeries* CUDA::FoldEngine::get_profiles ()
{
  return d_profiles;
}

void CUDA::FoldEngine::synch (dsp::PhaseSeries* output) try
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::synch this=" << this << endl;

  if (synchronized)
    return;

  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::synch output=" << output << endl;

  if (!transfer)
    transfer = new dsp::TransferCUDA;

  transfer->set_kind( hipMemcpyDeviceToHost );
  transfer->set_input( d_profiles );
  transfer->set_output( output );
  transfer->operate ();

  synchronized = true;
}
catch (Error& error)
{
  throw error += "CUDA::FoldEngine::synch";
}

void CUDA::FoldEngine::send_binplan ()
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::send_binplan ndat=" << ndat_fold 
         << " intervals=" << binplan.size() << endl;

  if (binplan.size() == 0)
    return;

  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::send_binplan"
            " first=" << binplan.front().ibin << 
            " last=" << binplan.back().ibin << endl;

  uint64_t mem_size = binplan.size() * sizeof(bin);

  if (binplan.size() > d_bin_size)
  {
    if (d_bin)
      hipFree (d_bin);

    hipMalloc ((void**)&d_bin, mem_size);
    d_bin_size = binplan.size();
  }
 
  // copy the kernel accross
  hipError_t error;
  error = hipMemcpy (d_bin, &(binplan[0]), mem_size, hipMemcpyHostToDevice);
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::FoldEngine::set_binplan",
                 "this=%x %s", this, hipGetErrorString (error));
}


__global__ void fold1bin (const float* in_base,
			     unsigned in_span,
			     float* out_base,
			     unsigned out_span,
			     unsigned ndim,
			     unsigned nbin,
			     unsigned binplan_size,
			     CUDA::bin* binplan)
{
  unsigned ibin = blockIdx.x * blockDim.x + threadIdx.x;

  if (ibin >= binplan_size)
    return;

  unsigned output_ibin = binplan[ibin].ibin;

  in_base += in_span * blockIdx.y + threadIdx.z;
  out_base += out_span * blockIdx.y + threadIdx.z;

  float total = 0;

  for (; ibin < binplan_size; ibin += nbin)
  {
    const float* input = in_base + binplan[ibin].offset * ndim;

    for (unsigned i=0; i < binplan[ibin].hits; i++)
      total += input[i*ndim];
  }

  out_base[ output_ibin * ndim ] += total;
}

std::ostream& operator<< (std::ostream& ostr, const dim3& v)
{
  return ostr << "(" << v.x << "," << v.y << "," << v.z << ")";
}

void CUDA::FoldEngine::fold ()
{
  setup ();
  send_binplan ();

  dim3 blockDim (128, 1, ndim);
  dim3 gridDim (folding_nbin/128, npol*nchan, 1);

#if 0
  cerr << "blockDim=" << blockDim << endl;
  cerr << "gridDim=" << gridDim << endl;
#endif

  fold1bin<<<gridDim,blockDim>>> (input, input_span, output, output_span,
                                  ndim, folding_nbin, binplan.size(), d_bin);

  // profile on the device is no longer synchronized with the one on the host
  synchronized = false;

  hipDeviceSynchronize ();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::FoldEngine::fold", 
                 hipGetErrorString (error));

}

